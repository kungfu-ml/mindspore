#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "backend/kernel_compiler/gpu/cuda_impl/adagrad_impl.cuh"

template <typename T>
__device__ __forceinline__ T SqrtFunc(T input) {
  return sqrt(input);
}

template <>
__device__ __forceinline__ half SqrtFunc(half input) {
  return hsqrt(input);
}

template <typename T>
__global__ void ApplyAdagradKernel(const size_t size,
                                   const bool update_slots,
                                   const T *learning_rate,
                                   const T *gradient,
                                   T *variable,
                                   T *accumulation) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    if (update_slots) {
      accumulation[i] += gradient[i] * gradient[i];
    }
    variable[i] -= learning_rate[0] * gradient[i] / SqrtFunc(accumulation[i]);
  }
}

template <typename T>
void ApplyAdagrad(const size_t size,
                  const bool update_slots,
                  const T *learning_rate,
                  const T *gradient,
                  T *variable,
                  T *accumulation,
                  hipStream_t hip_stream) {
  ApplyAdagradKernel<<< GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(
          size, update_slots, learning_rate, gradient, variable, accumulation);
}

template void ApplyAdagrad<float>(const size_t size,
                                  const bool update_slots,
                                  const float *learning_rate,
                                  const float *gradient,
                                  float *variable,
                                  float *accumulation,
                                  hipStream_t hip_stream);

template void ApplyAdagrad<half>(const size_t size,
                                 const bool update_slots,
                                 const half *learning_rate,
                                 const half *gradient,
                                 half *variable,
                                 half *accumulation,
                                 hipStream_t hip_stream);
